#include "hip/hip_runtime.h"
/* Copyright 2017-2021 The Simons Foundation. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <iostream>
#include <iomanip>
#include <math.h>
#include <tensorflow_nufft/third_party/cuda_samples/hip/hip_runtime_api.h>
#include <complex>
#include <hipfft/hipfft.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include "tensorflow_nufft/cc/kernels/finufft/gpu/cufinufft_eitherprec.h"
#include "tensorflow_nufft/cc/kernels/finufft/gpu/cuspreadinterp.h"
#include "tensorflow_nufft/cc/kernels/finufft/gpu/cudeconvolve.h"

using namespace std;
using namespace tensorflow;
using namespace tensorflow::nufft;


int CUFINUFFT2D1_EXEC(CUCPX* d_c, CUCPX* d_fk, Plan<GPUDevice, FLT>* d_plan)
/*  
  2D Type-1 NUFFT

  This function is called in "exec" stage (See ../cufinufft.cu).
  It includes (copied from doc in finufft library)
    Step 1: spread data to oversampled regular mesh using kernel
    Step 2: compute FFT on uniform mesh
    Step 3: deconvolve by division of each Fourier mode independently by the
            Fourier series coefficient of the kernel.

  Melody Shih 07/25/19		
*/
{
  int blksize;
  int ier;
  CUCPX* d_fkstart;
  CUCPX* d_cstart;
  for (int i=0; i*d_plan->options_.max_batch_size < d_plan->num_transforms_; i++) {
    blksize = min(d_plan->num_transforms_ - i*d_plan->options_.max_batch_size, 
      d_plan->options_.max_batch_size);
    d_cstart   = d_c + i*d_plan->options_.max_batch_size*d_plan->num_points_;
    d_fkstart  = d_fk + i*d_plan->options_.max_batch_size*d_plan->mode_count_;
    d_plan->c  = d_cstart;
    d_plan->fk = d_fkstart;

    checkCudaErrors(hipMemset(d_plan->fine_grid_data_,0,d_plan->options_.max_batch_size*
        d_plan->grid_size_ * sizeof(CUCPX)));

    // Step 1: Spread
    ier = CUSPREAD2D(d_plan,blksize);
    if (ier != 0 ) {
      printf("error: cuspread2d, method(%d)\n", d_plan->options_.spread_method);
      return ier;
    }

    // Step 2: FFT
    hipfftResult result = CUFFT_EX(
      d_plan->fft_plan_, d_plan->fine_grid_data_, d_plan->fine_grid_data_,
      static_cast<int>(d_plan->fft_direction_));
    if (result != HIPFFT_SUCCESS) {
      fprintf(stderr,"[%s] CUFFT_EX failed with error code: %d\n",__func__,result);
        return ERR_CUFFT;
    }

    // Step 3: deconvolve and shuffle
    CUDECONVOLVE2D(d_plan, blksize);
  }
  return ier;
}

int CUFINUFFT2D2_EXEC(CUCPX* d_c, CUCPX* d_fk, Plan<GPUDevice, FLT>* d_plan)
/*  
  2D Type-2 NUFFT

  This function is called in "exec" stage (See ../cufinufft.cu).
  It includes (copied from doc in finufft library)
    Step 1: deconvolve (amplify) each Fourier mode, dividing by kernel 
            Fourier coeff
    Step 2: compute FFT on uniform mesh
    Step 3: interpolate data to regular mesh

  Melody Shih 07/25/19
*/
{
  int blksize;
  int ier;
  CUCPX* d_fkstart;
  CUCPX* d_cstart;
  for (int i=0; i*d_plan->options_.max_batch_size < d_plan->num_transforms_; i++) {
    blksize = min(d_plan->num_transforms_ - i*d_plan->options_.max_batch_size, 
      d_plan->options_.max_batch_size);
    d_cstart  = d_c  + i*d_plan->options_.max_batch_size*d_plan->num_points_;
    d_fkstart = d_fk + i*d_plan->options_.max_batch_size*d_plan->mode_count_;

    d_plan->c = d_cstart;
    d_plan->fk = d_fkstart;

    // Step 1: amplify Fourier coeffs fk and copy into upsampled array fw
    CUDECONVOLVE2D(d_plan,blksize);

    // Step 2: FFT
    hipDeviceSynchronize();
    hipfftResult result = CUFFT_EX(
      d_plan->fft_plan_, d_plan->fine_grid_data_, d_plan->fine_grid_data_,
      static_cast<int>(d_plan->fft_direction_));
    if (result != HIPFFT_SUCCESS) {
      fprintf(stderr,"[%s] CUFFT_EX failed with error code: %d\n",__func__,result);
        return ERR_CUFFT;
    }

    // Step 3: deconvolve and shuffle
    ier = CUINTERP2D(d_plan, blksize);
    if (ier != 0 ) {
      printf("error: cuinterp2d, method(%d)\n", d_plan->options_.spread_method);
      return ier;
    }
  }
  return ier;
}

int CUFINUFFT2D_INTERP(CUCPX* d_c, CUCPX* d_fk, Plan<GPUDevice, FLT>* d_plan)
{
  assert(d_plan->spread_params_.spread_direction == SpreadDirection::INTERP);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  int blksize;
  int ier;
  int gridsize = d_plan->ms*d_plan->mt;
  CUCPX* d_fkstart;
  CUCPX* d_cstart;
  
  for (int i=0; i*d_plan->options_.max_batch_size < d_plan->num_transforms_; i++) {
    blksize = min(d_plan->num_transforms_ - i*d_plan->options_.max_batch_size, 
      d_plan->options_.max_batch_size);
    d_cstart  = d_c  + i*d_plan->options_.max_batch_size*d_plan->num_points_;
    d_fkstart = d_fk + i*d_plan->options_.max_batch_size*gridsize;

    d_plan->c = d_cstart;
    d_plan->fine_grid_data_ = d_fkstart;

    hipEventRecord(start);
    ier = CUINTERP2D(d_plan, blksize);
    if (ier != 0 ) {
      printf("error: cuinterp2d, method(%d)\n", d_plan->options_.spread_method);
      return ier;
    }
  }
  
  using namespace thrust::placeholders;
  thrust::device_ptr<FLT> dev_ptr((FLT*) d_c);
  thrust::transform(dev_ptr, dev_ptr + 2*d_plan->num_transforms_*d_plan->num_points_,
            dev_ptr, _1 * (FLT) d_plan->spread_params_.ES_scale); 
  
  return ier;
}

int CUFINUFFT2D_SPREAD(CUCPX* d_c, CUCPX* d_fk, Plan<GPUDevice, FLT>* d_plan)
{
  assert(d_plan->spread_params_.spread_direction == SpreadDirection::SPREAD);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  int blksize;
  int ier;
  int gridsize = d_plan->ms*d_plan->mt;
  CUCPX* d_fkstart;
  CUCPX* d_cstart;

  for (int i=0; i*d_plan->options_.max_batch_size < d_plan->num_transforms_; i++) {
    blksize = min(d_plan->num_transforms_ - i*d_plan->options_.max_batch_size, 
      d_plan->options_.max_batch_size);
    d_cstart   = d_c + i*d_plan->options_.max_batch_size*d_plan->num_points_;
    d_fkstart  = d_fk + i*d_plan->options_.max_batch_size*gridsize;
    
    d_plan->c  = d_cstart;
    d_plan->fine_grid_data_ = d_fkstart;

    hipEventRecord(start);
    ier = CUSPREAD2D(d_plan,blksize);
    if (ier != 0 ) {
      printf("error: cuspread2d, method(%d)\n", d_plan->options_.spread_method);
      return ier;
    }
  }

  using namespace thrust::placeholders;
  thrust::device_ptr<FLT> dev_ptr((FLT*) d_fk);
  thrust::transform(dev_ptr, dev_ptr + 2*d_plan->num_transforms_*gridsize,
            dev_ptr, _1 * (FLT) d_plan->spread_params_.ES_scale); 

  return ier;
}
