/* Copyright 2017-2021 The Simons Foundation. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <tensorflow_nufft/third_party/cuda_samples/hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

#include <hip/hip_complex.h>
#include "memtransfer.h"

using namespace std;
using namespace tensorflow;
using namespace tensorflow::nufft;



int ALLOCGPUMEM3D_NUPTS(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for gpu memory allocation in "setNUpts" stage.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options_.gpu_device_id);

	int M = d_plan->M;

	if (d_plan->sortidx ) checkCudaErrors(hipFree(d_plan->sortidx));
	if (d_plan->idxnupts) checkCudaErrors(hipFree(d_plan->idxnupts));

	switch (d_plan->options_.spread_method)
	{
		case SpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->spread_params_.sort_points == SortPoints::YES)
					checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
			}
			break;
		case SpreadMethod::SUBPROBLEM:
			{
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
			}
			break;
		case SpreadMethod::PAUL:
			{
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
			}
			break;
		case SpreadMethod::BLOCK_GATHER:
			{
				checkCudaErrors(hipMalloc(&d_plan->sortidx,M*sizeof(int)));
			}
			break;
		default:
			cerr << "err: invalid method" << endl;
	}

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);

	return 0;
}

