/* Copyright 2017-2021 The Simons Foundation. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <tensorflow_nufft/third_party/cuda_samples/hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

#include <hip/hip_complex.h>
#include "memtransfer.h"

using namespace std;
using namespace tensorflow;
using namespace tensorflow::nufft;


int ALLOCGPUMEM2D_NUPTS(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for gpu memory allocation in "setNUpts" stage.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options_.gpu_device_id);

	int M = d_plan->M;

	if (d_plan->sortidx ) checkCudaErrors(hipFree(d_plan->sortidx));
	if (d_plan->idxnupts) checkCudaErrors(hipFree(d_plan->idxnupts));

	switch(d_plan->options_.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options_.gpu_sort_points)
					checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
		case GpuSpreadMethod::PAUL:
			{
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
			}
			break;
		default:
			cerr<<"err: invalid method" << endl;
	}

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);

	return 0;
}


int ALLOCGPUMEM1D_NUPTS(Plan<GPUDevice, FLT>* d_plan)
{
	cerr<<"Not yet implemented"<<endl;
	return 1;
}


int ALLOCGPUMEM3D_NUPTS(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for gpu memory allocation in "setNUpts" stage.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options_.gpu_device_id);

	int M = d_plan->M;

	d_plan->byte_now=0;

	if (d_plan->sortidx ) checkCudaErrors(hipFree(d_plan->sortidx));
	if (d_plan->idxnupts) checkCudaErrors(hipFree(d_plan->idxnupts));

	switch (d_plan->options_.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options_.gpu_sort_points)
					checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
			{
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
			}
			break;
		case GpuSpreadMethod::BLOCK_GATHER:
			{
				checkCudaErrors(hipMalloc(&d_plan->sortidx,M*sizeof(int)));
			}
			break;
		default:
			cerr << "err: invalid method" << endl;
	}

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);

	return 0;
}

