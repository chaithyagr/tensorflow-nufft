/* Copyright 2017-2021 The Simons Foundation. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <tensorflow_nufft/third_party/cuda_samples/hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

#include <hip/hip_complex.h>
#include "memtransfer.h"

using namespace std;
using namespace tensorflow;
using namespace tensorflow::nufft;


int ALLOCGPUMEM2D_NUPTS(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for gpu memory allocation in "setNUpts" stage.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options_.gpu_device_id);

	int M = d_plan->M;

	if (d_plan->sortidx ) checkCudaErrors(hipFree(d_plan->sortidx));
	if (d_plan->idxnupts) checkCudaErrors(hipFree(d_plan->idxnupts));

	switch(d_plan->options_.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options_.gpu_sort_points)
					checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
		case GpuSpreadMethod::PAUL:
			{
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
			}
			break;
		default:
			cerr<<"err: invalid method" << endl;
	}

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);

	return 0;
}

void FREEGPUMEMORY2D(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for freeing gpu memory.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options_.gpu_device_id);

	if (!d_plan->options_.spread_interp_only) {
		checkCudaErrors(hipFree(d_plan->fw));
		checkCudaErrors(hipFree(d_plan->fwkerhalf1));
		checkCudaErrors(hipFree(d_plan->fwkerhalf2));
	}
	switch(d_plan->options_.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options_.gpu_sort_points) {
					checkCudaErrors(hipFree(d_plan->idxnupts));
					checkCudaErrors(hipFree(d_plan->sortidx));
					checkCudaErrors(hipFree(d_plan->binsize));
					checkCudaErrors(hipFree(d_plan->binstartpts));
				}else{
					checkCudaErrors(hipFree(d_plan->idxnupts));
				}
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
		case GpuSpreadMethod::PAUL:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->finegridsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
	}

	for (int i = 0; i < d_plan->options_.gpu_num_streams; i++)
		checkCudaErrors(hipStreamDestroy(d_plan->streams[i]));

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);
}

int ALLOCGPUMEM1D_NUPTS(Plan<GPUDevice, FLT>* d_plan)
{
	cerr<<"Not yet implemented"<<endl;
	return 1;
}
void FREEGPUMEMORY1D(Plan<GPUDevice, FLT>* d_plan)
{
	cerr<<"Not yet implemented"<<endl;
}


int ALLOCGPUMEM3D_NUPTS(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for gpu memory allocation in "setNUpts" stage.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options_.gpu_device_id);

	int M = d_plan->M;

	d_plan->byte_now=0;

	if (d_plan->sortidx ) checkCudaErrors(hipFree(d_plan->sortidx));
	if (d_plan->idxnupts) checkCudaErrors(hipFree(d_plan->idxnupts));

	switch (d_plan->options_.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options_.gpu_sort_points)
					checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
			{
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
			}
			break;
		case GpuSpreadMethod::BLOCK_GATHER:
			{
				checkCudaErrors(hipMalloc(&d_plan->sortidx,M*sizeof(int)));
			}
			break;
		default:
			cerr << "err: invalid method" << endl;
	}

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);

	return 0;
}
void FREEGPUMEMORY3D(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for freeing gpu memory.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options_.gpu_device_id);


	if (!d_plan->options_.spread_interp_only) {
		hipFree(d_plan->fw);
		hipFree(d_plan->fwkerhalf1);
		hipFree(d_plan->fwkerhalf2);
		hipFree(d_plan->fwkerhalf3);
	}

	switch (d_plan->options_.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options_.gpu_sort_points) {
					checkCudaErrors(hipFree(d_plan->idxnupts));
					checkCudaErrors(hipFree(d_plan->sortidx));
					checkCudaErrors(hipFree(d_plan->binsize));
					checkCudaErrors(hipFree(d_plan->binstartpts));
				}else{
					checkCudaErrors(hipFree(d_plan->idxnupts));
				}
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
		case GpuSpreadMethod::BLOCK_GATHER:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
	}

	for (int i = 0; i < d_plan->options_.gpu_num_streams; i++)
		checkCudaErrors(hipStreamDestroy(d_plan->streams[i]));

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);
}
