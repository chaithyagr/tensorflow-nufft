#include "hip/hip_runtime.h"
/* Copyright 2017-2021 The Simons Foundation. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <tensorflow_nufft/cc/kernels/finufft/gpu/contrib/cuda_samples/hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

#include <hip/hip_complex.h>
#include "memtransfer.h"

using namespace std;
using namespace tensorflow;
using namespace tensorflow::nufft;


int ALLOCGPUMEM2D_PLAN(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for gpu memory allocation in "plan" stage.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options.gpu_device_id);

	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int maxbatchsize = d_plan->maxbatchsize;

	d_plan->byte_now=0;
	// No extra memory is needed in nuptsdriven method (case 1)
	switch (d_plan->options.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options.gpu_sort_points) {
					int numbins[2];
					numbins[0] = ceil((FLT) nf1/d_plan->options.gpu_bin_size.x);
					numbins[1] = ceil((FLT) nf2/d_plan->options.gpu_bin_size.y);
					checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*sizeof(int)));
					checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*sizeof(int)));
				}
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
			{
				int numbins[2];
				numbins[0] = ceil((FLT) nf1/d_plan->options.gpu_bin_size.x);
				numbins[1] = ceil((FLT) nf2/d_plan->options.gpu_bin_size.y);
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,
						(numbins[0]*numbins[1]+1)*sizeof(int)));
			}
			break;
		case GpuSpreadMethod::PAUL:
			{
				int numbins[2];
				numbins[0] = ceil((FLT) nf1/d_plan->options.gpu_bin_size.x);
				numbins[1] = ceil((FLT) nf2/d_plan->options.gpu_bin_size.y);
				checkCudaErrors(hipMalloc(&d_plan->finegridsize,nf1*nf2*
						sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->fgstartpts,nf1*nf2*
						sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,
						(numbins[0]*numbins[1]+1)*sizeof(int)));
			}
			break;
		default:
			cerr << "err: invalid method " << endl;
	}

	if (!d_plan->options.spread_interp_only) {
		checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize*nf1*nf2*
				sizeof(CUCPX)));
		checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1,(nf1/2+1)*sizeof(FLT)));
		checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2,(nf2/2+1)*sizeof(FLT)));
	}

	hipStream_t* streams = (hipStream_t*) malloc(d_plan->options.gpu_num_streams*
		sizeof(hipStream_t));
	for (int i = 0; i < d_plan->options.gpu_num_streams; i++)
		checkCudaErrors(hipStreamCreate(&streams[i]));
	d_plan->streams = streams;

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);
	return 0;
}

int ALLOCGPUMEM2D_NUPTS(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for gpu memory allocation in "setNUpts" stage.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options.gpu_device_id);

	int M = d_plan->M;

	if (d_plan->sortidx ) checkCudaErrors(hipFree(d_plan->sortidx));
	if (d_plan->idxnupts) checkCudaErrors(hipFree(d_plan->idxnupts));

	switch(d_plan->options.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options.gpu_sort_points)
					checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
		case GpuSpreadMethod::PAUL:
			{
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
			}
			break;
		default:
			cerr<<"err: invalid method" << endl;
	}

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);

	return 0;
}

void FREEGPUMEMORY2D(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for freeing gpu memory.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options.gpu_device_id);

	if (!d_plan->options.spread_interp_only) {
		checkCudaErrors(hipFree(d_plan->fw));
		checkCudaErrors(hipFree(d_plan->fwkerhalf1));
		checkCudaErrors(hipFree(d_plan->fwkerhalf2));
	}
	switch(d_plan->options.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options.gpu_sort_points) {
					checkCudaErrors(hipFree(d_plan->idxnupts));
					checkCudaErrors(hipFree(d_plan->sortidx));
					checkCudaErrors(hipFree(d_plan->binsize));
					checkCudaErrors(hipFree(d_plan->binstartpts));
				}else{
					checkCudaErrors(hipFree(d_plan->idxnupts));
				}
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
		case GpuSpreadMethod::PAUL:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->finegridsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
	}

	for (int i = 0; i < d_plan->options.gpu_num_streams; i++)
		checkCudaErrors(hipStreamDestroy(d_plan->streams[i]));

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);
}

int ALLOCGPUMEM1D_PLAN(Plan<GPUDevice, FLT>* d_plan)
{
	cerr<<"Not yet implemented"<<endl;
	return 1;
}
int ALLOCGPUMEM1D_NUPTS(Plan<GPUDevice, FLT>* d_plan)
{
	cerr<<"Not yet implemented"<<endl;
	return 1;
}
void FREEGPUMEMORY1D(Plan<GPUDevice, FLT>* d_plan)
{
	cerr<<"Not yet implemented"<<endl;
}

int ALLOCGPUMEM3D_PLAN(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for gpu memory allocation in "plan" stage.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options.gpu_device_id);

	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int nf3 = d_plan->nf3;
	int maxbatchsize = d_plan->maxbatchsize;

	d_plan->byte_now=0;

	switch(d_plan->options.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options.gpu_sort_points) {
					int numbins[3];
					numbins[0] = ceil((FLT) nf1/d_plan->options.gpu_bin_size.x);
					numbins[1] = ceil((FLT) nf2/d_plan->options.gpu_bin_size.y);
					numbins[2] = ceil((FLT) nf3/d_plan->options.gpu_bin_size.z);
					checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*numbins[2]*sizeof(int)));
					checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*numbins[2]*sizeof(int)));
				}
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
			{
				int numbins[3];
				numbins[0] = ceil((FLT) nf1/d_plan->options.gpu_bin_size.x);
				numbins[1] = ceil((FLT) nf2/d_plan->options.gpu_bin_size.y);
				numbins[2] = ceil((FLT) nf3/d_plan->options.gpu_bin_size.z);
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
					numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
					numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
					numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,
					(numbins[0]*numbins[1]*numbins[2]+1)*sizeof(int)));
			}
			break;
		case GpuSpreadMethod::BLOCK_GATHER:
			{
				int numobins[3], numbins[3];
				int binsperobins[3];
				numobins[0] = ceil((FLT) nf1/d_plan->options.gpu_obin_size.x);
				numobins[1] = ceil((FLT) nf2/d_plan->options.gpu_obin_size.y);
				numobins[2] = ceil((FLT) nf3/d_plan->options.gpu_obin_size.z);

				binsperobins[0] = d_plan->options.gpu_obin_size.x/
					d_plan->options.gpu_bin_size.x;
				binsperobins[1] = d_plan->options.gpu_obin_size.y/
					d_plan->options.gpu_bin_size.y;
				binsperobins[2] = d_plan->options.gpu_obin_size.z/
					d_plan->options.gpu_bin_size.z;

				numbins[0] = numobins[0]*(binsperobins[0]+2);
				numbins[1] = numobins[1]*(binsperobins[1]+2);
				numbins[2] = numobins[2]*(binsperobins[2]+2);

				checkCudaErrors(hipMalloc(&d_plan->numsubprob,
					numobins[0]*numobins[1]*numobins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,
					numbins[0]*numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,
					(numbins[0]*numbins[1]*numbins[2]+1)*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,(numobins[0]
					*numobins[1]*numobins[2]+1)*sizeof(int)));
			}
			break;
		default:
			cerr << "err: invalid method" << endl;
	}

	if (!d_plan->options.spread_interp_only) {
		checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize*nf1*nf2*nf3*
			sizeof(CUCPX)));
		checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1,(nf1/2+1)*sizeof(FLT)));
		checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2,(nf2/2+1)*sizeof(FLT)));
		checkCudaErrors(hipMalloc(&d_plan->fwkerhalf3,(nf3/2+1)*sizeof(FLT)));
	}

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);

	return 0;
}

int ALLOCGPUMEM3D_NUPTS(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for gpu memory allocation in "setNUpts" stage.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options.gpu_device_id);

	int M = d_plan->M;

	d_plan->byte_now=0;

	if (d_plan->sortidx ) checkCudaErrors(hipFree(d_plan->sortidx));
	if (d_plan->idxnupts) checkCudaErrors(hipFree(d_plan->idxnupts));

	switch (d_plan->options.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options.gpu_sort_points)
					checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
			{
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
			}
			break;
		case GpuSpreadMethod::BLOCK_GATHER:
			{
				checkCudaErrors(hipMalloc(&d_plan->sortidx,M*sizeof(int)));
			}
			break;
		default:
			cerr << "err: invalid method" << endl;
	}

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);

	return 0;
}
void FREEGPUMEMORY3D(Plan<GPUDevice, FLT>* d_plan)
/*
	wrapper for freeing gpu memory.

	Melody Shih 07/25/19
*/
{
        // Mult-GPU support: set the CUDA Device ID:
        int orig_gpu_device_id;
        hipGetDevice(& orig_gpu_device_id);
        hipSetDevice(d_plan->options.gpu_device_id);


	if (!d_plan->options.spread_interp_only) {
		hipFree(d_plan->fw);
		hipFree(d_plan->fwkerhalf1);
		hipFree(d_plan->fwkerhalf2);
		hipFree(d_plan->fwkerhalf3);
	}

	switch (d_plan->options.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				if (d_plan->options.gpu_sort_points) {
					checkCudaErrors(hipFree(d_plan->idxnupts));
					checkCudaErrors(hipFree(d_plan->sortidx));
					checkCudaErrors(hipFree(d_plan->binsize));
					checkCudaErrors(hipFree(d_plan->binstartpts));
				}else{
					checkCudaErrors(hipFree(d_plan->idxnupts));
				}
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
		case GpuSpreadMethod::BLOCK_GATHER:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
	}

	for (int i = 0; i < d_plan->options.gpu_num_streams; i++)
		checkCudaErrors(hipStreamDestroy(d_plan->streams[i]));

        // Multi-GPU support: reset the device ID
        hipSetDevice(orig_gpu_device_id);
}
