#include "hip/hip_runtime.h"
/* Copyright 2017-2021 The Simons Foundation. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <tensorflow_nufft/cc/kernels/finufft/gpu/contrib/cuda_samples/hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

#include <hip/hip_complex.h>
#include "tensorflow_nufft/cc/kernels/finufft/gpu/cuspreadinterp.h"
#include "tensorflow_nufft/cc/kernels/finufft/gpu/memtransfer.h"
#include <profile.h>
#include "tensorflow_nufft/cc/kernels/nufft_options.h"
#include "tensorflow_nufft/cc/kernels/nufft_plan.h"

using namespace std;
using namespace tensorflow;
using namespace tensorflow::nufft;


int CUFINUFFT_INTERP2D(int nf1, int nf2, CUCPX* d_fw, int M, 
	FLT *d_kx, FLT *d_ky, CUCPX *d_c, Plan<GPUDevice, FLT>* d_plan)
/*
	This c function is written for only doing 2D interpolation. See 
	test/interp2d_test.cu for usage.

	Melody Shih 07/25/19
	not allocate,transfer and free memories on gpu. Shih 09/24/20
*/
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	d_plan->nf1 = nf1;
	d_plan->nf2 = nf2;
	d_plan->M = M;
	d_plan->maxbatchsize = 1;

	d_plan->kx = d_kx;
	d_plan->ky = d_ky;
	d_plan->c  = d_c;
	d_plan->fw = d_fw;

	int ier;
	hipEventRecord(start);
	ier = ALLOCGPUMEM2D_PLAN(d_plan);
	ier = ALLOCGPUMEM2D_NUPTS(d_plan);
	if (d_plan->options.gpu_spread_method == GpuSpreadMethod::NUPTS_DRIVEN) {
		ier = CUSPREAD2D_NUPTSDRIVEN_PROP(nf1,nf2,M,d_plan);
		if (ier != 0 ) {
			printf("error: cuspread2d_subprob_prop, method(%d)\n", 
				d_plan->options.gpu_spread_method);
			return ier;
		}
	}
	if (d_plan->options.gpu_spread_method == GpuSpreadMethod::SUBPROBLEM) {
		ier = CUSPREAD2D_SUBPROB_PROP(nf1,nf2,M,d_plan);
		if (ier != 0 ) {
			printf("error: cuspread2d_subprob_prop, method(%d)\n", 
				d_plan->options.gpu_spread_method);
			return ier;
		}
	}
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Obtain Interp Prop\t %.3g ms\n", d_plan->options.gpu_spread_method, 
		milliseconds);
#endif
	hipEventRecord(start);
	ier = CUINTERP2D(d_plan,1);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Interp (%d)\t\t %.3g ms\n", d_plan->options.gpu_spread_method, 
		milliseconds);
#endif
	hipEventRecord(start);
	FREEGPUMEMORY2D(d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
	return ier;
}

int CUINTERP2D(Plan<GPUDevice, FLT>* d_plan, int blksize)
/*
	A wrapper for different interpolation methods. 

	Methods available:
	(1) Non-uniform points driven
	(2) Subproblem

	Melody Shih 07/25/19
*/
{
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int M = d_plan->M;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	switch (d_plan->options.gpu_spread_method)
	{
		case GpuSpreadMethod::NUPTS_DRIVEN:
			{
				hipEventRecord(start);
				{
					PROFILE_CUDA_GROUP("Spreading", 6);
					ier = CUINTERP2D_NUPTSDRIVEN(nf1, nf2, M, d_plan, blksize);
					if (ier != 0 ) {
						cout<<"error: cnufftspread2d_gpu_nuptsdriven"<<endl;
						return 1;
					}
				}
			}
			break;
		case GpuSpreadMethod::SUBPROBLEM:
			{
				hipEventRecord(start);
				ier = CUINTERP2D_SUBPROB(nf1, nf2, M, d_plan, blksize);
				if (ier != 0 ) {
					cout<<"error: cuinterp2d_subprob"<<endl;
					return 1;
				}
			}
			break;
		default:
			cout<<"error: incorrect method, should be 1 or 2"<<endl;
			return 2;
	}
#ifdef SPREADTIME
	float milliseconds;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Interp " << milliseconds <<" ms"<<endl;
#endif
	return ier;
}

int CUINTERP2D_NUPTSDRIVEN(int nf1, int nf2, int M, Plan<GPUDevice, FLT>* d_plan,
	int blksize)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=d_plan->spopts.nspread;   // psi's support in terms of number of cells
	FLT es_c=d_plan->spopts.ES_c;
	FLT es_beta=d_plan->spopts.ES_beta;
	FLT sigma = d_plan->options.upsampling_factor;
	int pirange=d_plan->spopts.pirange;
	int *d_idxnupts=d_plan->idxnupts;

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	threadsPerBlock.x = 32;
	threadsPerBlock.y = 1;
	blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = 1;

	hipEventRecord(start);
	if (d_plan->options.kernel_evaluation_method == KernelEvaluationMethod::HORNER) {
		for (int t=0; t<blksize; t++) {
			Interp_2d_NUptsdriven_Horner<<<blocks, threadsPerBlock>>>(d_kx, 
				d_ky, d_c+t*M, d_fw+t*nf1*nf2, M, ns, nf1, nf2, sigma, 
				d_idxnupts, pirange);
		}
	}else{
		for (int t=0; t<blksize; t++) {
			Interp_2d_NUptsdriven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, 
				d_c+t*M, d_fw+t*nf1*nf2, M, ns, nf1, nf2, es_c, es_beta, 
				d_idxnupts, pirange);
		}
	}
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Interp_2d_NUptsdriven (%d)\t%.3g ms\n", 
		milliseconds, d_plan->options.kernel_evaluation_method);
#endif
	return 0;
}

int CUINTERP2D_SUBPROB(int nf1, int nf2, int M, Plan<GPUDevice, FLT>* d_plan,
	int blksize)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ns=d_plan->spopts.nspread;   // psi's support in terms of number of cells
	FLT es_c=d_plan->spopts.ES_c;
	FLT es_beta=d_plan->spopts.ES_beta;
	int maxsubprobsize=d_plan->options.gpu_max_subproblem_size;

	// assume that bin_size_x > ns/2;
	int bin_size_x=d_plan->options.gpu_bin_size.x;
	int bin_size_y=d_plan->options.gpu_bin_size.y;
	int numbins[2];
	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
#ifdef INFO
	cout<<"[info  ] Dividing the uniform grids to bin size["
		<<d_plan->options.gpu_bin_size.x<<"x"<<d_plan->options.gpu_bin_size.y<<"]"<<endl;
	cout<<"[info  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	int *d_binsize = d_plan->binsize;
	int *d_binstartpts = d_plan->binstartpts;
	int *d_numsubprob = d_plan->numsubprob;
	int *d_subprobstartpts = d_plan->subprobstartpts;
	int *d_idxnupts = d_plan->idxnupts;
	int *d_subprob_to_bin = d_plan->subprob_to_bin;
	int totalnumsubprob=d_plan->totalnumsubprob;
	int pirange=d_plan->spopts.pirange;

	FLT sigma=d_plan->options.upsampling_factor;
	hipEventRecord(start);
	size_t sharedplanorysize = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*
		ceil(ns/2.0))*sizeof(CUCPX);
	if (sharedplanorysize > 49152) {
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}

	if (d_plan->options.kernel_evaluation_method == KernelEvaluationMethod::HORNER) {
		for (int t=0; t<blksize; t++) {
			Interp_2d_Subprob_Horner<<<totalnumsubprob, 256, sharedplanorysize>>>(
					d_kx, d_ky, d_c+t*M,
					d_fw+t*nf1*nf2, M, ns, nf1, nf2, sigma,
					d_binstartpts, d_binsize,
					bin_size_x, bin_size_y,
					d_subprob_to_bin, d_subprobstartpts,
					d_numsubprob, maxsubprobsize,
					numbins[0], numbins[1], d_idxnupts, pirange);
		}
	} else {
		for (int t=0; t<blksize; t++) {
			Interp_2d_Subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(
					d_kx, d_ky, d_c+t*M,
					d_fw+t*nf1*nf2, M, ns, nf1, nf2,
					es_c, es_beta, sigma,
					d_binstartpts, d_binsize,
					bin_size_x, bin_size_y,
					d_subprob_to_bin, d_subprobstartpts,
					d_numsubprob, maxsubprobsize,
					numbins[0], numbins[1], d_idxnupts, pirange);
		}
	}
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Interp_2d_Subprob (%d)\t\t%.3g ms\n", 
		milliseconds, d_plan->options.kernel_evaluation_method);
#endif
	return 0;
}
