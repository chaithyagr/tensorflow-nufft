/* Copyright 2017-2021 The Simons Foundation. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <tensorflow_nufft/third_party/cuda_samples/hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

#include <hip/hip_complex.h>
#include "tensorflow_nufft/cc/kernels/finufft/gpu/cuspreadinterp.h"
#include "tensorflow_nufft/cc/kernels/finufft/gpu/memtransfer.h"
#include <profile.h>
#include "tensorflow_nufft/cc/kernels/nufft_options.h"
#include "tensorflow_nufft/cc/kernels/nufft_plan.h"

using namespace std;
using namespace tensorflow;
using namespace tensorflow::nufft;


int CUINTERP2D(Plan<GPUDevice, FLT>* d_plan, int blksize)
/*
	A wrapper for different interpolation methods. 

	Methods available:
	(1) Non-uniform points driven
	(2) Subproblem

	Melody Shih 07/25/19
*/
{
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int M = d_plan->M;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	switch (d_plan->options_.spread_method)
	{
		case SpreadMethod::NUPTS_DRIVEN:
			{
				hipEventRecord(start);
				{
					PROFILE_CUDA_GROUP("Spreading", 6);
					ier = CUINTERP2D_NUPTSDRIVEN(d_plan, blksize);
					if (ier != 0 ) {
						cout<<"error: cnufftspread2d_gpu_nuptsdriven"<<endl;
						return 1;
					}
				}
			}
			break;
		case SpreadMethod::SUBPROBLEM:
			{
				hipEventRecord(start);
				ier = CUINTERP2D_SUBPROB(d_plan, blksize);
				if (ier != 0 ) {
					cout<<"error: cuinterp2d_subprob"<<endl;
					return 1;
				}
			}
			break;
		default:
			cout<<"error: incorrect method, should be 1 or 2"<<endl;
			return 2;
	}
#ifdef SPREADTIME
	float milliseconds;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Interp " << milliseconds <<" ms"<<endl;
#endif
	return ier;
}

