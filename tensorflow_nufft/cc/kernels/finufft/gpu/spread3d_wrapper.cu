#include "hip/hip_runtime.h"
/* Copyright 2017-2021 The Simons Foundation. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <tensorflow_nufft/third_party/cuda_samples/hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>
#include <assert.h>

#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <hip/hip_complex.h>
#include "tensorflow_nufft/cc/kernels/finufft/gpu/cuspreadinterp.h"
#include "tensorflow_nufft/cc/kernels/finufft/gpu/memtransfer.h"
#include "tensorflow_nufft/cc/kernels/finufft/gpu/precision_independent.h"

using namespace std;
using namespace tensorflow;
using namespace tensorflow::nufft;


int CUSPREAD3D(Plan<GPUDevice, FLT>* d_plan, int blksize)
/*
  A wrapper for different spreading methods. 

  Methods available:
  (1) Non-uniform points driven
  (2) Subproblem
  (4) Block gather

  Melody Shih 07/25/19
*/
{
  int nf1 = d_plan->nf1;
  int nf2 = d_plan->nf2;
  int nf3 = d_plan->nf3;
  int M = d_plan->M;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int ier = 0;
  switch(d_plan->options_.spread_method)
  {
    case SpreadMethod::NUPTS_DRIVEN:
      {
        hipEventRecord(start);
        ier = CUSPREAD2D_NUPTSDRIVEN(d_plan, blksize);
        if (ier != 0 ) {
          cout<<"error: cnufftspread3d_gpu_subprob"<<endl;
          return 1;
        }
      }
      break;
    case SpreadMethod::SUBPROBLEM:
      {
        hipEventRecord(start);
        ier = CUSPREAD2D_SUBPROB(d_plan, blksize);
        if (ier != 0 ) {
          cout<<"error: cnufftspread3d_gpu_subprob"<<endl;
          return 1;
        }
      }
      break;
    case SpreadMethod::BLOCK_GATHER:
      {
        hipEventRecord(start);
        ier = CUSPREAD3D_BLOCKGATHER(nf1, nf2, nf3, M, d_plan, blksize);
        if (ier != 0 ) {
          cout<<"error: cnufftspread3d_gpu_subprob"<<endl;
          return 1;
        }
      }
      break;
    default:
      cerr<<"error: incorrect method, should be 1,2,4"<<endl;
      return 2;
  }
  return ier;
}

int CUSPREAD3D_BLOCKGATHER_PROP(int nf1, int nf2, int nf3, int M, 
  Plan<GPUDevice, FLT>* d_plan)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  dim3 threadsPerBlock;
  dim3 blocks;

  int pirange = d_plan->spread_params_.pirange;

  int maxsubprobsize=d_plan->options_.gpu_max_subproblem_size;
  int o_bin_size_x = d_plan->options_.gpu_obin_size.x;
  int o_bin_size_y = d_plan->options_.gpu_obin_size.y;
  int o_bin_size_z = d_plan->options_.gpu_obin_size.z;
  
  int numobins[3];
  if (nf1 % o_bin_size_x != 0 ||
    nf2 % o_bin_size_y != 0 ||
    nf3 % o_bin_size_z != 0) {
    cout<<"error: mod(nf1, options.gpu_obin_size.x) != 0"<<endl;
    cout<<"       mod(nf2, options.gpu_obin_size.y) != 0"<<endl;
    cout<<"       mod(nf3, options.gpu_obin_size.z) != 0"<<endl;
    cout<<"error: (nf1, nf2, nf3) = ("<<nf1<<", "<<nf2<<", "<<nf3<<")"<<endl;
    cout<<"error: (obinsizex, obinsizey, obinsizez) = ("
      <<o_bin_size_x<<", "<<o_bin_size_y<<", "<<o_bin_size_z<<")"<<endl;
    return 1;
  }

  numobins[0] = ceil((FLT) nf1/o_bin_size_x);
  numobins[1] = ceil((FLT) nf2/o_bin_size_y);
  numobins[2] = ceil((FLT) nf3/o_bin_size_z);

  int bin_size_x=d_plan->options_.gpu_bin_size.x;
  int bin_size_y=d_plan->options_.gpu_bin_size.y;
  int bin_size_z=d_plan->options_.gpu_bin_size.z;
  if (o_bin_size_x % bin_size_x != 0 ||
    o_bin_size_y % bin_size_y != 0 ||
    o_bin_size_z % bin_size_z != 0) {
    cout<<"error: mod(ops.gpu_obin_size.x, options.gpu_bin_size.x) != 0"<<endl;
    cout<<"       mod(ops.gpu_obin_size.y, options.gpu_bin_size.y) != 0"<<endl;
    cout<<"       mod(ops.gpu_obin_size.z, options.gpu_bin_size.z) != 0"<<endl;
    cout<<"error: (binsizex, binsizey, binsizez) = ("
      <<bin_size_x<<", "<<bin_size_y<<", "<<bin_size_z<<")"<<endl;
    cout<<"error: (obinsizex, obinsizey, obinsizez) = ("
      <<o_bin_size_x<<", "<<o_bin_size_y<<", "<<o_bin_size_z<<")"<<endl;
    return 1;
  }

  int binsperobinx, binsperobiny, binsperobinz;
  int numbins[3];
  binsperobinx = o_bin_size_x/bin_size_x+2;
  binsperobiny = o_bin_size_y/bin_size_y+2;
  binsperobinz = o_bin_size_z/bin_size_z+2;
  numbins[0] = numobins[0]*(binsperobinx);
  numbins[1] = numobins[1]*(binsperobiny);
  numbins[2] = numobins[2]*(binsperobinz);
#ifdef DEBUG
  cout<<"[debug ] Dividing the uniform grids to bin size["
    <<d_plan->options_.gpu_bin_size.x<<"x"<<d_plan->options_.gpu_bin_size.y<<"x"<<
    d_plan->options_.gpu_bin_size.z<<"]"<<endl;
  cout<<"[debug ] numobins = ["<<numobins[0]<<"x"<<numobins[1]<<"x"<<
    numobins[2]<<"]"<<endl;
  cout<<"[debug ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"x"<<
    numbins[2]<<"]"<<endl;
#endif

  FLT*   d_kx = d_plan->kx;
  FLT*   d_ky = d_plan->ky;
  FLT*   d_kz = d_plan->kz;

#ifdef DEBUG
  FLT *h_kx, *h_ky, *h_kz;
  h_kx = (FLT*)malloc(M*sizeof(FLT));
  h_ky = (FLT*)malloc(M*sizeof(FLT));
  h_kz = (FLT*)malloc(M*sizeof(FLT));

  checkCudaErrors(hipMemcpy(h_kx,d_kx,M*sizeof(FLT),hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_ky,d_ky,M*sizeof(FLT),hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_kz,d_kz,M*sizeof(FLT),hipMemcpyDeviceToHost));
  for (int i=0; i<M; i++) {
    cout<<"[debug ] ";
    cout <<"("<<setw(3)<<h_kx[i]<<","<<setw(3)<<h_ky[i]<<","<<h_kz[i]<<")"
      <<endl;
  }
#endif
  int *d_binsize = d_plan->binsize;
  int *d_sortidx = d_plan->sortidx;
  int *d_binstartpts = d_plan->binstartpts;
  int *d_numsubprob = d_plan->numsubprob;
  void*d_temp_storage = NULL;
  int *d_idxnupts = NULL;
  int *d_subprobstartpts = d_plan->subprobstartpts;
  int *d_subprob_to_bin = NULL;

  // Synchronize device before we start. This is essential! Otherwise the
  // next kernel could read the wrong (kx, ky, kz) values.
  checkCudaErrors(hipDeviceSynchronize());

  hipEventRecord(start);
  checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*numbins[2]*
    sizeof(int)));
  LocateNUptstoBins_ghost<<<(M+1024-1)/1024, 1024>>>(M,bin_size_x,
    bin_size_y,bin_size_z,numobins[0],numobins[1],numobins[2],binsperobinx,
    binsperobiny, binsperobinz,d_binsize,d_kx,
    d_ky,d_kz,d_sortidx,pirange,nf1,nf2,nf3);

#ifdef SPREADTIME
  float milliseconds = 0;
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel LocateNUptstoBins_ghost \t\t%.3g ms\n",
    milliseconds);
#endif
#ifdef DEBUG
  int *h_binsize;// For debug
  h_binsize     = (int*)malloc(numbins[0]*numbins[1]*numbins[2]*sizeof(int));
  checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*
    numbins[2]*sizeof(int),hipMemcpyDeviceToHost));
  cout<<"[debug ] bin size:"<<endl;
  for (int k=0; k<numbins[2]; k++) {
    cout<<"[debug ]"<<endl;
    for (int j=0; j<numbins[1]; j++) {
      if (j%binsperobinx == 0 && j!=0)
        cout<<"[debug ] -----------------"<<endl;
      cout<<"[debug ] ";
      for (int i=0; i<numbins[0]; i++) {
        if (i%binsperobinx == 0 && i!=0)
          cout<<"|";
        if (i!=0) cout<<" ";
        int binidx = CalcGlobalIdx(i,j,k,numobins[0],numobins[1],
          numobins[2],binsperobinx,binsperobiny,binsperobinz);
        cout<<h_binsize[binidx];
      }
      cout<<endl;
    }
  }
  cout<<"[debug ] ---------------------------------------------------"<<endl;
#endif
#ifdef DEBUG
  int *h_sortidx;
  h_sortidx = (int*)malloc(M*sizeof(int));

  checkCudaErrors(hipMemcpy(h_sortidx,d_sortidx,M*sizeof(int),
    hipMemcpyDeviceToHost));
  for (int i=0; i<M; i++) {
    cout <<"[debug ] point["<<setw(3)<<i<<"]="<<setw(3)<<h_sortidx[i]<<endl;
  }
#endif
  hipEventRecord(start);
  threadsPerBlock.x=8;
  threadsPerBlock.y=8;
  threadsPerBlock.z=8;

  blocks.x = (threadsPerBlock.x+numbins[0]-1)/threadsPerBlock.x;
  blocks.y = (threadsPerBlock.y+numbins[1]-1)/threadsPerBlock.y;
  blocks.z = (threadsPerBlock.z+numbins[2]-1)/threadsPerBlock.z;

  FillGhostBins<<<blocks, threadsPerBlock>>>(binsperobinx, binsperobiny,
    binsperobinz, numobins[0], numobins[1], numobins[2], d_binsize);
#ifdef SPREADTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel FillGhostBins \t\t\t%.3g ms\n",
    milliseconds);
#endif
#ifdef DEBUG
  checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*
    numbins[2]*sizeof(int),hipMemcpyDeviceToHost));
  cout<<"[debug ] Filled ghost bins:"<<endl;
  for (int k=0; k<numbins[2]; k++) {
    cout<<"[debug ] "<<endl;
    cout<<"[debug ] "<<endl;
    for (int j=0; j<numbins[1]; j++) {
      if (j%binsperobinx == 0 && j!=0)
        cout<<"[debug ] -----------------"<<endl;
      cout<<"[debug ] ";
      for (int i=0; i<numbins[0]; i++) {
        if (i%binsperobinx == 0 && i!=0)
        cout<<"|";
        int binidx = CalcGlobalIdx(i,j,k,numobins[0],numobins[1],
          numobins[2],binsperobinx,binsperobiny,binsperobinz);
        if (i!=0) cout<<" ";
        cout<<h_binsize[binidx];
      }
      cout<<endl;
    }
  }
  cout<<"[debug ] ---------------------------------------------------"<<endl;
#endif
  hipEventRecord(start);
  int n=numbins[0]*numbins[1]*numbins[2];
  thrust::device_ptr<int> d_ptr(d_binsize);
  thrust::device_ptr<int> d_result(d_binstartpts+1);
  thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
  checkCudaErrors(hipMemset(d_binstartpts,0,sizeof(int)));
#ifdef SPREADTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel BinStartPts_3d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
  int *h_binstartpts;
  h_binstartpts = (int*)malloc((numbins[0]*numbins[1]*numbins[2])*sizeof(int));
  checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,(numbins[0]*
    numbins[1]*numbins[2])*sizeof(int),hipMemcpyDeviceToHost));
  cout<<"[debug ] Result of scan bin_size array:"<<endl;
  for (int k=0; k<numbins[2]; k++) {
    cout<<"[debug ] "<<endl;
    for (int j=0; j<numbins[1]; j++) {
      cout<<"[debug ] ";
      for (int i=0; i<numbins[0]; i++) {
        if (i!=0) cout<<" ";
        int binidx = CalcGlobalIdx(i,j,k,numobins[0],numobins[1],
          numobins[2],binsperobinx,binsperobiny,binsperobinz);
        cout<<h_binstartpts[binidx];
      }
      cout<<endl;
    }
  }
  cout<<"[debug ] ----------------------------------------------------"<<endl;
#endif
  hipEventRecord(start);
  int totalNUpts;
  checkCudaErrors(hipMemcpy(&totalNUpts,&d_binstartpts[n],
    sizeof(int),hipMemcpyDeviceToHost));
  checkCudaErrors(hipMalloc(&d_idxnupts,totalNUpts*sizeof(int)));
#ifdef DEBUG
  checkCudaErrors(hipMemset(d_idxnupts,-1,totalNUpts*sizeof(int)));
#endif
  hipEventRecord(start);
  CalcInvertofGlobalSortIdx_ghost<<<(M+1024-1)/1024,1024>>>(M,bin_size_x,
    bin_size_y,bin_size_z,numobins[0],numobins[1],numobins[2],binsperobinx,
    binsperobiny,binsperobinz,d_binstartpts,d_sortidx,d_kx,d_ky,d_kz,
    d_idxnupts,pirange,nf1,nf2,nf3);
#ifdef SPREADTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel CalcInvertofGlobalIdx_ghost \t%.3g ms\n",
    milliseconds);
#endif
#ifdef DEBUG
  int *h_idxnupts;
  h_idxnupts = (int*)malloc(totalNUpts*sizeof(int));
  checkCudaErrors(hipMemcpy(h_idxnupts,d_idxnupts,totalNUpts*sizeof(int),
    hipMemcpyDeviceToHost));
  int pts = 0;
  for (int b=0; b<numbins[0]*numbins[1]*numbins[1]; b++) {
    if (h_binsize[b] > 0)
      cout <<"[debug ] Bin "<<b<<endl;
    for (int i=h_binstartpts[b]; i<h_binstartpts[b]+h_binsize[b]; i++) {
      cout <<"[debug ] NUpts-index= "<< h_idxnupts[i]<<endl;
      pts++;
    }
  }
  cout<<"[debug ] totalpts = "<<pts<<endl;
#endif
  hipEventRecord(start);
  threadsPerBlock.x=2;
  threadsPerBlock.y=2;
  threadsPerBlock.z=2;

  blocks.x = (threadsPerBlock.x+numbins[0]-1)/threadsPerBlock.x;
  blocks.y = (threadsPerBlock.y+numbins[1]-1)/threadsPerBlock.y;
  blocks.z = (threadsPerBlock.z+numbins[2]-1)/threadsPerBlock.z;

  GhostBinPtsIdx<<<blocks, threadsPerBlock>>>(binsperobinx, binsperobiny,
    binsperobinz, numobins[0], numobins[1], numobins[2], d_binsize,
    d_idxnupts, d_binstartpts, M);
        if (d_plan->idxnupts != NULL) hipFree(d_plan->idxnupts);
  d_plan->idxnupts = d_idxnupts;
#ifdef SPREADTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel GhostBinPtsIdx \t\t\t%.3g ms\n",
    milliseconds);
#endif
#ifdef DEBUG
  checkCudaErrors(hipMemcpy(h_idxnupts,d_idxnupts,totalNUpts*sizeof(int),
    hipMemcpyDeviceToHost));
  pts = 0;
  for (int b=0; b<numbins[0]*numbins[1]*numbins[1]; b++) {
    if (h_binsize[b] > 0)
      cout <<"[debug ] Bin "<<b<<endl;
    for (int i=h_binstartpts[b]; i<h_binstartpts[b]+h_binsize[b]; i++) {
      cout <<"[debug ] NUpts-index= "<< h_idxnupts[i]<<endl;
      pts++;
    }
  }
  cout<<"[debug ] totalpts = "<<pts<<endl;
  free(h_idxnupts);
  free(h_binstartpts);
  free(h_binsize);
#endif

  /* --------------------------------------------- */
  //        Determining Subproblem properties      //
  /* --------------------------------------------- */
  hipEventRecord(start);
  n = numobins[0]*numobins[1]*numobins[2];
  hipEventRecord(start);
  CalcSubProb_3d_v1<<<(n+1024-1)/1024, 1024>>>(binsperobinx, binsperobiny,
    binsperobinz, d_binsize, d_numsubprob, maxsubprobsize, numobins[0]*
    numobins[1]*numobins[2]);
#ifdef SPREADTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel CalcSubProb_3d_v1\t\t%.3g ms\n",
    milliseconds);
#endif
#ifdef DEBUG
  int* h_numsubprob;
  h_numsubprob = (int*) malloc(n*sizeof(int));
  checkCudaErrors(hipMemcpy(h_numsubprob,d_numsubprob,numobins[0]*numobins[1]*
    numobins[2]*sizeof(int),hipMemcpyDeviceToHost));
  for (int k=0; k<numobins[2]; k++) {
    cout<<"[debug ] "<<endl;
    for (int j=0; j<numobins[1]; j++) {
      cout<<"[debug ] ";
      for (int i=0; i<numobins[0]; i++) {
        if (i!=0) cout<<" ";
        cout <<"s["<<setw(1)<<i<<","<<setw(1)<<j<<","<<setw(1)<<k
          <<"]= "<<setw(3)<<h_numsubprob[i+j*numobins[0]+k*
          numobins[1]*numobins[2]];
      }
      cout<<endl;
    }
  }
  free(h_numsubprob);
#endif
  hipEventRecord(start);
  n = numobins[0]*numobins[1]*numobins[2];
  d_ptr    = thrust::device_pointer_cast(d_numsubprob);
  d_result = thrust::device_pointer_cast(d_subprobstartpts+1);
  thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
  checkCudaErrors(hipMemset(d_subprobstartpts,0,sizeof(int)));
#ifdef SPREADTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tScan  numsubprob\t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
  printf("[debug ] Subproblem start points\n");
  int* h_subprobstartpts;
  h_subprobstartpts = (int*) malloc((n+1)*sizeof(int));
  checkCudaErrors(hipMemcpy(h_subprobstartpts,d_subprobstartpts,(numobins[0]*
    numobins[1]*numobins[2]+1)*sizeof(int),hipMemcpyDeviceToHost));
  for (int k=0; k<numobins[2]; k++) {
    if (k!=0)
      cout<<"[debug ] "<<endl;
    for (int j=0; j<numobins[1]; j++) {
      cout<<"[debug ] ";
      for (int i=0; i<numobins[0]; i++) {
        if (i!=0) cout<<" ";
        cout <<"s["<<setw(1)<<i<<","<<setw(1)<<j<<","<<setw(1)<<k
          <<"]= "<<setw(3)<<h_subprobstartpts[i+j*numobins[0]+k*
          numobins[1]*numobins[2]];
      }
      cout<<endl;
    }
  }
  printf("[debug ] Total number of subproblems (%d) = %d\n", n,
    h_subprobstartpts[n]);
  free(h_subprobstartpts);
  cout<<"[debug ] ---------------------------------------------------"<<endl;
#endif
  hipEventRecord(start);
  int totalnumsubprob;
  checkCudaErrors(hipMemcpy(&totalnumsubprob,&d_subprobstartpts[n],
    sizeof(int),hipMemcpyDeviceToHost));
  checkCudaErrors(hipMalloc(&d_subprob_to_bin,totalnumsubprob*sizeof(int)));
  MapBintoSubProb_3d_v1<<<(n+1024-1)/1024, 1024>>>(d_subprob_to_bin,
    d_subprobstartpts,d_numsubprob,n);
  assert(d_subprob_to_bin != NULL);
        if (d_plan->subprob_to_bin != NULL) hipFree(d_plan->subprob_to_bin);
  d_plan->subprob_to_bin   = d_subprob_to_bin;
  d_plan->totalnumsubprob  = totalnumsubprob;
#ifdef SPREADTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel Subproblem to Bin map\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
  printf("[debug ] Map Subproblem to Bins\n");
  int* h_subprob_to_bin;
  h_subprob_to_bin   = (int*) malloc((totalnumsubprob)*sizeof(int));
  checkCudaErrors(hipMemcpy(h_subprob_to_bin,d_subprob_to_bin,
    (totalnumsubprob)*sizeof(int),hipMemcpyDeviceToHost));
  for (int j=0; j<totalnumsubprob; j++) {
    cout<<"[debug ] ";
    cout <<"s["<<j<<"] = "<<setw(2)<<"b["<<h_subprob_to_bin[j]<<"]";
    cout<<endl;
  }
  free(h_subprob_to_bin);
#endif
  hipFree(d_temp_storage);

  return 0;
}

int CUSPREAD3D_BLOCKGATHER(int nf1, int nf2, int nf3, int M, 
  Plan<GPUDevice, FLT>* d_plan, int blksize)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int ns=d_plan->spread_params_.nspread; 
  FLT es_c=d_plan->spread_params_.ES_c;
  FLT es_beta=d_plan->spread_params_.ES_beta;
  FLT sigma=d_plan->spread_params_.upsampling_factor;
  int pirange=d_plan->spread_params_.pirange;
  int maxsubprobsize=d_plan->options_.gpu_max_subproblem_size;

  int obin_size_x=d_plan->options_.gpu_obin_size.x;
  int obin_size_y=d_plan->options_.gpu_obin_size.y;
  int obin_size_z=d_plan->options_.gpu_obin_size.z;
  int bin_size_x=d_plan->options_.gpu_bin_size.x;
  int bin_size_y=d_plan->options_.gpu_bin_size.y;
  int bin_size_z=d_plan->options_.gpu_bin_size.z;
  int numobins[3];
  numobins[0] = ceil((FLT) nf1/obin_size_x);
  numobins[1] = ceil((FLT) nf2/obin_size_y);
  numobins[2] = ceil((FLT) nf3/obin_size_z);

  int binsperobinx, binsperobiny, binsperobinz;
  binsperobinx = obin_size_x/bin_size_x+2;
  binsperobiny = obin_size_y/bin_size_y+2;
  binsperobinz = obin_size_z/bin_size_z+2;
#ifdef INFO
  cout<<"[info  ] Dividing the uniform grids to bin size["
    <<obin_size_x<<"x"<<obin_size_y<<"x"<<obin_size_z<<"]"<<endl;
  cout<<"[info  ] numbins = ["<<numobins[0]<<"x"<<numobins[1]<<"x"<<
    numobins[2]<<"]"<<endl;
  cout<<"[info  ] ns = "<< ns<<endl;
#endif

  FLT* d_kx = d_plan->kx;
  FLT* d_ky = d_plan->ky;
  FLT* d_kz = d_plan->kz;
  CUCPX* d_c = d_plan->c;
  CUCPX* d_fw = d_plan->fine_grid_data_;

  int *d_binstartpts = d_plan->binstartpts;
  int *d_subprobstartpts = d_plan->subprobstartpts;
  int *d_idxnupts = d_plan->idxnupts;

  int totalnumsubprob=d_plan->totalnumsubprob;
  int *d_subprob_to_bin = d_plan->subprob_to_bin;

  hipEventRecord(start);
  for (int t=0; t<blksize; t++) {
    if (d_plan->options_.kernel_evaluation_method == KernelEvaluationMethod::HORNER) {
      size_t sharedplanorysize = obin_size_x*obin_size_y*obin_size_z
        *sizeof(CUCPX);
      if (sharedplanorysize > 49152) {
        cout<<"error: not enough shared memory"<<endl;
        return 1;
      }
      Spread_3d_BlockGather_Horner<<<totalnumsubprob, 64, sharedplanorysize
        >>>(d_kx, d_ky, d_kz, d_c+t*M, d_fw+t*nf1*nf2*nf3, M, ns,
          nf1, nf2, nf3, es_c, es_beta, sigma, d_binstartpts,
          obin_size_x, obin_size_y, obin_size_z,
          binsperobinx*binsperobiny*binsperobinz,d_subprob_to_bin,
          d_subprobstartpts, maxsubprobsize, numobins[0], numobins[1],
          numobins[2], d_idxnupts,pirange);
    }else{
      size_t sharedplanorysize = obin_size_x*obin_size_y*obin_size_z
        *sizeof(CUCPX);
      if (sharedplanorysize > 49152) {
        cout<<"error: not enough shared memory"<<endl;
        return 1;
      }
      Spread_3d_BlockGather<<<totalnumsubprob, 64, sharedplanorysize>>>(
          d_kx, d_ky, d_kz, d_c+t*M, d_fw+t*nf1*nf2*nf3, M, ns,
          nf1, nf2, nf3, es_c, es_beta, sigma, d_binstartpts,
          obin_size_x, obin_size_y, obin_size_z,
          binsperobinx*binsperobiny*binsperobinz,d_subprob_to_bin,
          d_subprobstartpts, maxsubprobsize, numobins[0], numobins[1],
          numobins[2], d_idxnupts,pirange);
    }
  }
#ifdef SPREADTIME
      float milliseconds = 0;
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds, start, stop);
      printf("[time  ] \tKernel Spread_3d_BlockGather (%d)\t%.3g ms\n",
        milliseconds, d_plan->options_.kernel_evaluation_method);
#endif
  return 0;
}

int CUSPREAD3D_SUBPROB_PROP(int nf1, int nf2, int nf3, int M, 
  Plan<GPUDevice, FLT>* d_plan)
{

  int maxsubprobsize=d_plan->options_.gpu_max_subproblem_size;
  int bin_size_x=d_plan->options_.gpu_bin_size.x;
  int bin_size_y=d_plan->options_.gpu_bin_size.y;
  int bin_size_z=d_plan->options_.gpu_bin_size.z;
  if (bin_size_x < 0 || bin_size_y < 0 || bin_size_z < 0) {
    cout<<"error: invalid binsize (binsizex, binsizey, binsizez) = (";
    cout<<bin_size_x<<","<<bin_size_y<<","<<bin_size_z<<")"<<endl;
    return 1; 
  }

  int numbins[3];
  numbins[0] = ceil((FLT) nf1/bin_size_x);
  numbins[1] = ceil((FLT) nf2/bin_size_y);
  numbins[2] = ceil((FLT) nf3/bin_size_z);

  FLT*   d_kx = d_plan->kx;
  FLT*   d_ky = d_plan->ky;
  FLT*   d_kz = d_plan->kz;

  int *d_binsize = d_plan->binsize;
  int *d_binstartpts = d_plan->binstartpts;
  int *d_sortidx = d_plan->sortidx;
  int *d_numsubprob = d_plan->numsubprob;
  int *d_subprobstartpts = d_plan->subprobstartpts;
  int *d_idxnupts = d_plan->idxnupts;

  int *d_subprob_to_bin = NULL;
  void *d_temp_storage = NULL;
  int pirange = d_plan->spread_params_.pirange;

  // Synchronize device before we start. This is essential! Otherwise the
  // next kernel could read the wrong (kx, ky, kz) values.
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*numbins[2]*
    sizeof(int)));
  CalcBinSizeNoGhost3DKernel<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,nf3,bin_size_x,
    bin_size_y,bin_size_z,numbins[0],numbins[1],numbins[2],d_binsize,d_kx,
    d_ky,d_kz,d_sortidx,pirange);


  int n=numbins[0]*numbins[1]*numbins[2];
  thrust::device_ptr<int> d_ptr(d_binsize);
  thrust::device_ptr<int> d_result(d_binstartpts);
  thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);


  CalcInvertofGlobalSortIdx3DKernel<<<(M+1024-1)/1024,1024>>>(M,bin_size_x,
    bin_size_y,bin_size_z,numbins[0],numbins[1],numbins[2], d_binstartpts,
    d_sortidx,d_kx,d_ky,d_kz,d_idxnupts,pirange,nf1,nf2,nf3);

  /* --------------------------------------------- */
  //        Determining Subproblem properties      //
  /* --------------------------------------------- */
  CalcSubProb_3d_v2<<<(M+1024-1)/1024, 1024>>>(d_binsize,d_numsubprob,
      maxsubprobsize,numbins[0]*numbins[1]*numbins[2]);

  d_ptr    = thrust::device_pointer_cast(d_numsubprob);
  d_result = thrust::device_pointer_cast(d_subprobstartpts+1);
  thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
  checkCudaErrors(hipMemset(d_subprobstartpts,0,sizeof(int)));

  int totalnumsubprob;
  checkCudaErrors(hipMemcpy(&totalnumsubprob,&d_subprobstartpts[n],
    sizeof(int),hipMemcpyDeviceToHost));
  checkCudaErrors(hipMalloc(&d_subprob_to_bin,totalnumsubprob*sizeof(int)));
  MapBintoSubProb_3d_v2<<<(numbins[0]*numbins[1]+1024-1)/1024, 1024>>>(
    d_subprob_to_bin,d_subprobstartpts,d_numsubprob,numbins[0]*numbins[1]*
    numbins[2]);
  assert(d_subprob_to_bin != NULL);
        if (d_plan->subprob_to_bin != NULL) hipFree(d_plan->subprob_to_bin);
  d_plan->subprob_to_bin = d_subprob_to_bin;
  assert(d_plan->subprob_to_bin != NULL);
  d_plan->totalnumsubprob = totalnumsubprob;

  hipFree(d_temp_storage);

  return 0;
}
