#include "hip/hip_runtime.h"
/* Copyright 2017-2021 The Simons Foundation. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <tensorflow_nufft/third_party/cuda_samples/hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>
#include <assert.h>

#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <hip/hip_complex.h>
#include "tensorflow_nufft/cc/kernels/finufft/gpu/cuspreadinterp.h"
#include "tensorflow_nufft/cc/kernels/finufft/gpu/memtransfer.h"

using namespace std;
using namespace tensorflow;
using namespace tensorflow::nufft;


int CUSPREAD2D(Plan<GPUDevice, FLT>* d_plan, int blksize)
/*
  A wrapper for different spreading methods.

  Methods available:
  (1) Non-uniform points driven
  (2) Subproblem
  (3) Paul

  Melody Shih 07/25/19
*/
{
  int nf1 = d_plan->nf1;
  int nf2 = d_plan->nf2;
  int M = d_plan->M;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int ier;
  switch(d_plan->options_.spread_method)
  {
    case SpreadMethod::NUPTS_DRIVEN:
      {
        hipEventRecord(start);
        ier = CUSPREAD2D_NUPTSDRIVEN(nf1, nf2, M, d_plan, blksize);
        if (ier != 0 ) {
          cout<<"error: cnufftspread2d_gpu_nuptsdriven"<<endl;
          return 1;
        }
      }
      break;
    case SpreadMethod::SUBPROBLEM:
      {
        hipEventRecord(start);
        ier = CUSPREAD2D_SUBPROB(nf1, nf2, M, d_plan, blksize);
        if (ier != 0 ) {
          cout<<"error: cnufftspread2d_gpu_subprob"<<endl;
          return 1;
        }
      }
      break;
    case SpreadMethod::PAUL:
      {
        hipEventRecord(start);
        ier = CUSPREAD2D_PAUL(nf1, nf2, M, d_plan, blksize);
        if (ier != 0 ) {
          cout<<"error: cnufftspread2d_gpu_paul"<<endl;
          return 1;
        }
      }
      break;
    default:
      cout<<"error: incorrect method, should be 1,2,3"<<endl;
      return 2;
  }
#ifdef SPREADTIME
  float milliseconds = 0;
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  cout<<"[time  ]"<< " Spread " << milliseconds <<" ms"<<endl;
#endif
  return ier;
}

int CUSPREAD2D_NUPTSDRIVEN_PROP(int nf1, int nf2, int M, Plan<GPUDevice, FLT>* d_plan)
{
  if (d_plan->spread_params_.sort_points == SortPoints::YES) {

    int bin_size[2];
    bin_size[0] = d_plan->options_.gpu_bin_size.x;
    bin_size[1] = d_plan->options_.gpu_bin_size.y;
    if (bin_size[0] < 0 || bin_size[1] < 0) {
      cout << "error: invalid binsize (binsizex, binsizey) = (";
      cout << bin_size[0] << "," << bin_size[1] << ")" << endl;
      return 1; 
    }

    int numbins[2];
    numbins[0] = ceil((FLT) nf1 / bin_size[0]);
    numbins[1] = ceil((FLT) nf2 / bin_size[1]);

    FLT*   d_kx = d_plan->kx;
    FLT*   d_ky = d_plan->ky;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_sortidx = d_plan->sortidx;
    int *d_idxnupts = d_plan->idxnupts;

    int pirange = d_plan->spread_params_.pirange;

    // Synchronize device before we start. This is essential! Otherwise the
    // next kernel could read the wrong (kx, ky, kz) values.
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*
      sizeof(int)));
    CalcBinSize_noghost_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, nf1, nf2,
      bin_size[0], bin_size[1], numbins[0], numbins[1],
      d_binsize, d_kx, d_ky, d_sortidx, pirange);

    int n=numbins[0]*numbins[1];
    thrust::device_ptr<int> d_ptr(d_binsize);
    thrust::device_ptr<int> d_result(d_binstartpts);
    thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);

    CalcInvertofGlobalSortIdx_2d<<<(M + 1024 - 1) / 1024, 1024>>>(
      M, bin_size[0], bin_size[1], numbins[0], numbins[1],
      d_binstartpts, d_sortidx, d_kx, d_ky,
      d_idxnupts, pirange, nf1, nf2);

  }else{
    int *d_idxnupts = d_plan->idxnupts;

    TrivialGlobalSortIdx_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, d_idxnupts);
  }
  return 0;
}

int CUSPREAD2D_NUPTSDRIVEN(int nf1, int nf2, int M, Plan<GPUDevice, FLT>* d_plan,
  int blksize)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  dim3 threadsPerBlock;
  dim3 blocks;

  int ns=d_plan->spread_params_.nspread;   // psi's support in terms of number of cells
  int pirange=d_plan->spread_params_.pirange;
  int *d_idxnupts=d_plan->idxnupts;
  FLT es_c=d_plan->spread_params_.ES_c;
  FLT es_beta=d_plan->spread_params_.ES_beta;
  FLT sigma=d_plan->spread_params_.upsampling_factor;

  FLT* d_kx = d_plan->kx;
  FLT* d_ky = d_plan->ky;
  CUCPX* d_c = d_plan->c;
  CUCPX* d_fw = d_plan->fine_grid_data_;

  threadsPerBlock.x = 16;
  threadsPerBlock.y = 1;
  blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
  blocks.y = 1;
  hipEventRecord(start);
  if (d_plan->options_.kernel_evaluation_method == KernelEvaluationMethod::HORNER) {
    for (int t=0; t<blksize; t++) {
      Spread_2d_NUptsdriven_Horner<<<blocks, threadsPerBlock>>>(d_kx,
        d_ky, d_c+t*M, d_fw+t*nf1*nf2, M, ns, nf1, nf2, sigma,
        d_idxnupts, pirange);
    }
  } else {
    for (int t=0; t<blksize; t++) {
      Spread_2d_NUptsdriven<<<blocks, threadsPerBlock>>>(d_kx, d_ky,
        d_c+t*M, d_fw+t*nf1*nf2, M, ns, nf1, nf2, es_c, es_beta,
        d_idxnupts, pirange);
    }
  }

#ifdef SPREADTIME
  float milliseconds = 0;
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel Spread_2d_NUptsdriven (%d)\t%.3g ms\n",
    milliseconds, d_plan->options_.kernel_evaluation_method);
#endif
  return 0;
}
int CUSPREAD2D_SUBPROB_PROP(int nf1, int nf2, int M, Plan<GPUDevice, FLT>* d_plan)
/*
  This function determines the properties for spreading that are independent
  of the strength of the nodes,  only relates to the locations of the nodes,
  which only needs to be done once.
*/
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int maxsubprobsize=d_plan->options_.gpu_max_subproblem_size;
  int bin_size_x=d_plan->options_.gpu_bin_size.x;
  int bin_size_y=d_plan->options_.gpu_bin_size.y;
  if (bin_size_x < 0 || bin_size_y < 0) {
    cout<<"error: invalid binsize (binsizex, binsizey) = (";
    cout<<bin_size_x<<","<<bin_size_y<<")"<<endl;
    return 1; 
  }
  int numbins[2];
  numbins[0] = ceil((FLT) nf1/bin_size_x);
  numbins[1] = ceil((FLT) nf2/bin_size_y);
#ifdef DEBUG
  cout<<"[debug  ] Dividing the uniform grids to bin size["
    <<d_plan->options_.gpu_bin_size.x<<"x"<<d_plan->options_.gpu_bin_size.y<<"]"<<endl;
  cout<<"[debug  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif

  FLT*   d_kx = d_plan->kx;
  FLT*   d_ky = d_plan->ky;

#ifdef DEBUG
  FLT *h_kx;
  FLT *h_ky;
  h_kx = (FLT*)malloc(M*sizeof(FLT));
  h_ky = (FLT*)malloc(M*sizeof(FLT));

  checkCudaErrors(hipMemcpy(h_kx,d_kx,M*sizeof(FLT),hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_ky,d_ky,M*sizeof(FLT),hipMemcpyDeviceToHost));
  for (int i=0; i<M; i++) {
    cout<<"[debug ]";
    cout <<"("<<setw(3)<<h_kx[i]<<","<<setw(3)<<h_ky[i]<<")"<<endl;
  }
#endif
  int *d_binsize = d_plan->binsize;
  int *d_binstartpts = d_plan->binstartpts;
  int *d_sortidx = d_plan->sortidx;
  int *d_numsubprob = d_plan->numsubprob;
  int *d_subprobstartpts = d_plan->subprobstartpts;
  int *d_idxnupts = d_plan->idxnupts;

  int *d_subprob_to_bin = NULL;

  int pirange=d_plan->spread_params_.pirange;

  // Synchronize device before we start. This is essential! Otherwise the
  // next kernel could read the wrong (kx, ky, kz) values.
  checkCudaErrors(hipDeviceSynchronize());

  hipEventRecord(start);
  checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*sizeof(int)));
  CalcBinSize_noghost_2d<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,bin_size_x,
    bin_size_y,numbins[0],numbins[1],d_binsize,d_kx,d_ky,d_sortidx,pirange);
#ifdef SPREADTIME
  float milliseconds = 0;
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n",
    milliseconds);
#endif
#ifdef DEBUG
  int *h_binsize;// For debug
  h_binsize     = (int*)malloc(numbins[0]*numbins[1]*sizeof(int));
  checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*
    sizeof(int),hipMemcpyDeviceToHost));
  cout<<"[debug ] bin size:"<<endl;
  for (int j=0; j<numbins[1]; j++) {
    cout<<"[debug ] ";
    for (int i=0; i<numbins[0]; i++) {
      if (i!=0) cout<<" ";
      cout <<" bin["<<setw(3)<<i<<","<<setw(3)<<j<<"]="<<
        h_binsize[i+j*numbins[0]];
    }
    cout<<endl;
  }
  free(h_binsize);
  cout<<"[debug ] ----------------------------------------------------"<<endl;
#endif
#ifdef DEBUG
  int *h_sortidx;
  h_sortidx = (int*)malloc(M*sizeof(int));
  checkCudaErrors(hipMemcpy(h_sortidx,d_sortidx,M*sizeof(int),
    hipMemcpyDeviceToHost));
  cout<<"[debug ]";
  for (int i=0; i<M; i++) {
    cout <<"[debug] point["<<setw(3)<<i<<"]="<<setw(3)<<h_sortidx[i]<<endl;
  }

#endif

  hipEventRecord(start);
  int n=numbins[0]*numbins[1];
  thrust::device_ptr<int> d_ptr(d_binsize);
  thrust::device_ptr<int> d_result(d_binstartpts);
  thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
#ifdef SPREADTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
  int *h_binstartpts;
  h_binstartpts = (int*)malloc((numbins[0]*numbins[1])*sizeof(int));
  checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,
        (numbins[0]*numbins[1])*sizeof(int),
        hipMemcpyDeviceToHost));
  cout<<"[debug ] Result of scan bin_size array:"<<endl;
  for (int j=0; j<numbins[1]; j++) {
    cout<<"[debug ] ";
    for (int i=0; i<numbins[0]; i++) {
      if (i!=0) cout<<" ";
      cout <<"bin["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)
        <<h_binstartpts[i+j*numbins[0]];
    }
    cout<<endl;
  }
  free(h_binstartpts);
  cout<<"[debug ] ---------------------------------------------------"<<endl;
#endif
  hipEventRecord(start);
  CalcInvertofGlobalSortIdx_2d<<<(M+1024-1)/1024,1024>>>(M,bin_size_x,
    bin_size_y,numbins[0],numbins[1],d_binstartpts,d_sortidx,d_kx,d_ky,
    d_idxnupts,pirange,nf1,nf2);
#ifdef DEBUG
  int *h_idxnupts;
  h_idxnupts = (int*)malloc(M*sizeof(int));
  checkCudaErrors(hipMemcpy(h_idxnupts,d_idxnupts,M*sizeof(int),
        hipMemcpyDeviceToHost));
  for (int i=0; i<M; i++) {
    cout <<"[debug ] idx="<< h_idxnupts[i]<<endl;
  }
  free(h_idxnupts);
#endif
  hipEventRecord(start);
  CalcSubProb_2d<<<(M+1024-1)/1024, 1024>>>(d_binsize,d_numsubprob,
    maxsubprobsize,numbins[0]*numbins[1]);
#ifdef SPREADTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel CalcSubProb_2d\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
  int* h_numsubprob;
  h_numsubprob = (int*) malloc(n*sizeof(int));
  checkCudaErrors(hipMemcpy(h_numsubprob,d_numsubprob,numbins[0]*numbins[1]*
        sizeof(int),hipMemcpyDeviceToHost));
  for (int j=0; j<numbins[1]; j++) {
    cout<<"[debug ] ";
    for (int i=0; i<numbins[0]; i++) {
      if (i!=0) cout<<" ";
      cout <<"nsub["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<
        h_numsubprob[i+j*numbins[0]];
    }
    cout<<endl;
  }
  free(h_numsubprob);
#endif
  d_ptr    = thrust::device_pointer_cast(d_numsubprob);
  d_result = thrust::device_pointer_cast(d_subprobstartpts+1);
  thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
  checkCudaErrors(hipMemset(d_subprobstartpts,0,sizeof(int)));
#ifdef SPREADTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel Scan Subprob array\t\t%.3g ms\n", milliseconds);
#endif

#ifdef DEBUG
  printf("[debug ] Subproblem start points\n");
  int* h_subprobstartpts;
  h_subprobstartpts = (int*) malloc((n+1)*sizeof(int));
  checkCudaErrors(hipMemcpy(h_subprobstartpts,d_subprobstartpts,
        (n+1)*sizeof(int),hipMemcpyDeviceToHost));
  for (int j=0; j<numbins[1]; j++) {
    cout<<"[debug ] ";
    for (int i=0; i<numbins[0]; i++) {
      if (i!=0) cout<<" ";
      cout <<"nsub["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<
        h_subprobstartpts[i+j*numbins[0]];
    }
    cout<<endl;
  }
  printf("[debug ] Total number of subproblems = %d\n", h_subprobstartpts[n]);
  free(h_subprobstartpts);
#endif
  hipEventRecord(start);
  int totalnumsubprob;
  checkCudaErrors(hipMemcpy(&totalnumsubprob,&d_subprobstartpts[n],
    sizeof(int),hipMemcpyDeviceToHost));
  checkCudaErrors(hipMalloc(&d_subprob_to_bin,totalnumsubprob*sizeof(int)));
  MapBintoSubProb_2d<<<(numbins[0]*numbins[1]+1024-1)/1024, 1024>>>(
      d_subprob_to_bin,d_subprobstartpts,d_numsubprob,numbins[0]*numbins[1]);
  assert(d_subprob_to_bin != NULL);
        if (d_plan->subprob_to_bin != NULL) hipFree(d_plan->subprob_to_bin);
  d_plan->subprob_to_bin = d_subprob_to_bin;
  assert(d_plan->subprob_to_bin != NULL);
  d_plan->totalnumsubprob = totalnumsubprob;
#ifdef DEBUG
  printf("[debug ] Map Subproblem to Bins\n");
  int* h_subprob_to_bin;
  h_subprob_to_bin = (int*) malloc((totalnumsubprob)*sizeof(int));
  checkCudaErrors(hipMemcpy(h_subprob_to_bin,d_subprob_to_bin,
        (totalnumsubprob)*sizeof(int),hipMemcpyDeviceToHost));
  for (int j=0; j<totalnumsubprob; j++) {
    cout<<"[debug ] ";
    cout <<"nsub["<<j<<"] = "<<setw(2)<<h_subprob_to_bin[j];
    cout<<endl;
  }
  free(h_subprob_to_bin);
#endif
#ifdef SPREADTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel Subproblem to Bin map\t\t%.3g ms\n", milliseconds);
#endif
  return 0;
}

int CUSPREAD2D_SUBPROB(int nf1, int nf2, int M, Plan<GPUDevice, FLT>* d_plan,
  int blksize)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int ns=d_plan->spread_params_.nspread;// psi's support in terms of number of cells
  FLT es_c=d_plan->spread_params_.ES_c;
  FLT es_beta=d_plan->spread_params_.ES_beta;
  int maxsubprobsize=d_plan->options_.gpu_max_subproblem_size;

  // assume that bin_size_x > ns/2;
  int bin_size_x=d_plan->options_.gpu_bin_size.x;
  int bin_size_y=d_plan->options_.gpu_bin_size.y;
  int numbins[2];
  numbins[0] = ceil((FLT) nf1/bin_size_x);
  numbins[1] = ceil((FLT) nf2/bin_size_y);
#ifdef INFO
  cout<<"[info  ] Dividing the uniform grids to bin size["
    <<d_plan->options_.gpu_bin_size.x<<"x"<<d_plan->options_.gpu_bin_size.y<<"]"<<endl;
  cout<<"[info  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif

  FLT* d_kx = d_plan->kx;
  FLT* d_ky = d_plan->ky;
  CUCPX* d_c = d_plan->c;
  CUCPX* d_fw = d_plan->fine_grid_data_;

  int *d_binsize = d_plan->binsize;
  int *d_binstartpts = d_plan->binstartpts;
  int *d_numsubprob = d_plan->numsubprob;
  int *d_subprobstartpts = d_plan->subprobstartpts;
  int *d_idxnupts = d_plan->idxnupts;

  int totalnumsubprob=d_plan->totalnumsubprob;
  int *d_subprob_to_bin = d_plan->subprob_to_bin;

  int pirange=d_plan->spread_params_.pirange;

  FLT sigma=d_plan->options_.upsampling_factor;
  hipEventRecord(start);

  size_t sharedplanorysize = (bin_size_x+2*(int)ceil(ns/2.0))*
                 (bin_size_y+2*(int)ceil(ns/2.0))*
                 sizeof(CUCPX);
  if (sharedplanorysize > 49152) {
    cout<<"error: not enough shared memory"<<endl;
    return 1;
  }

  if (d_plan->options_.kernel_evaluation_method == KernelEvaluationMethod::HORNER) {
    for (int t=0; t<blksize; t++) {
      Spread_2d_Subprob_Horner<<<totalnumsubprob, 256,
        sharedplanorysize>>>(d_kx, d_ky, d_c+t*M, d_fw+t*nf1*nf2, M,
        ns, nf1, nf2, sigma, d_binstartpts, d_binsize, bin_size_x,
        bin_size_y, d_subprob_to_bin, d_subprobstartpts,
        d_numsubprob, maxsubprobsize,numbins[0],numbins[1],
        d_idxnupts, pirange);
    }
  }else{
    for (int t=0; t<blksize; t++) {
      Spread_2d_Subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(
        d_kx, d_ky, d_c+t*M, d_fw+t*nf1*nf2, M, ns, nf1, nf2,
        es_c, es_beta, sigma,d_binstartpts, d_binsize, bin_size_x,
        bin_size_y, d_subprob_to_bin, d_subprobstartpts,
        d_numsubprob, maxsubprobsize, numbins[0], numbins[1],
        d_idxnupts, pirange);
    }
  }
#ifdef SPREADTIME
  float milliseconds = 0;
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("[time  ] \tKernel Spread_2d_Subprob (%d)\t\t%.3g ms\n",
    milliseconds, d_plan->options_.kernel_evaluation_method);
#endif
  return 0;
}
